#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include<stdio.h>
#include<conio.h>

__device__  long long int mod(int base,int exponent,int den)
{
	unsigned int a=(base%den)*(base%den);
	unsigned long long int ret=1;
	float size=(float)exponent/2;
	if(exponent==0)
	{
		return base%den;
	}
	else
	{
		while(1)
		{
			if(size>0.5)
			{
				ret=(ret*a)%den;
				size=size-1.0;
			}
			else if(size==0.5)
			{
			ret=(ret*(base%den))%den;
			break;
			}
			else
			{
				break;
			}
		}
	return ret;
	}
}

__global__ void rsa(int * num,int *key,int *den,unsigned int * result)
{
int i=threadIdx.x;
int temp;

 if(i<3)
 {   
	temp=mod(num[i],*key,*den);
	atomicExch(&result[i],temp);
 }
}

void loadDefaultImage(char *loc_exec)
{

    printf("Reading image: lena.pgm\n");
    const char *image_filename = "lena.pgm";
    char *image_path = sdkFindFilePath(image_filename, loc_exec);

    if (image_path == NULL)
    {
        printf("Failed to read image file: <%s>\n", image_filename);
        exit(EXIT_FAILURE);
    }

    initializeData(image_path);
    free(image_path);
}

 int main()
 {
 int  num[3]={16,5,4};
 int key=5;
 int den=35;
 int devcount;
 hipGetDeviceCount(&devcount);
 printf("%d CUDA devices found",devcount);
 
 if(devcount>0)
 {
 hipSetDevice(1);
 printf("\nEnter the 8 digit word:");
 for(int i=0;i<3;i++)
 {
	 printf("\n.");
	 scanf("%d",&num[i]);
 }
 printf("\nEnter key parameter 1:");
 scanf("%d",&key);
 printf("\nEnter key parameter 2:");
 scanf("%d",&den);
 
 int *dev_num,*dev_key,*dev_den;
 unsigned int *dev_res;
 unsigned  int res[3]={1,1,1};
 dim3 grid(1,1,1);
 dim3 block(3,3,1);
 
 hipMalloc( (void **)&dev_num, 3*sizeof(int));
 hipMalloc( (void **)&dev_key,sizeof(int));
 hipMalloc( (void **)&dev_den, sizeof(int));
 hipMalloc( (void **)&dev_res, 3*sizeof(unsigned int));     



      switch (key)
    {
        case 27:
        case 'q':
        case 'Q':
            printf("Shutting down...\n");
            exit(EXIT_SUCCESS);
            break;

        case '-':
            imageScale -= 0.1f;
            printf("brightness = %4.2f\n", imageScale);
            break;

        case '=':
            imageScale += 0.1f;
            printf("brightness = %4.2f\n", imageScale);
            break;

        case 'i':
        case 'I':
            g_SobelDisplayMode = SOBELDISPLAY_IMAGE;
            sprintf(temp, "CUDA Edge Detection (%s)", filterMode[g_SobelDisplayMode]);
            glutSetWindowTitle(temp);
            break;

        case 's':
        case 'S':
            g_SobelDisplayMode = SOBELDISPLAY_SOBELSHARED;
            sprintf(temp, "CUDA Edge Detection (%s)", filterMode[g_SobelDisplayMode]);
            glutSetWindowTitle(temp);
            break;

        case 't':
        case 'T':
            g_SobelDisplayMode = SOBELDISPLAY_SOBELTEX;
            sprintf(temp, "CUDA Edge Detection (%s)", filterMode[g_SobelDisplayMode]);
            glutSetWindowTitle(temp);
            break;

        default:
            break;
    }
}
 hipMemcpy(dev_num,num,3*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_key,&key,sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_den,&den,sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_res,res,3*sizeof(unsigned int),hipMemcpyHostToDevice);    
 rsa<<<grid,block>>>(dev_num,dev_key,dev_den,dev_res);
hipMemcpy(res,dev_res,3*sizeof(unsigned int),hipMemcpyDeviceToHost);
hipFree(dev_num);
hipFree(dev_key);
hipFree(dev_den);
hipFree(dev_res);
for(int i=0;i<3;i++)
{
	printf("\n%d",res[i]);
}
 }
getch();
 return 0;
 }     
    
 